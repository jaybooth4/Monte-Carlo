// GPU monte Carlo Simulation to calculate the value of pi


#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdio.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#define N 100
#define NUMTHREADS 32
#define NUMBLOCKS 32
#define NUMDARTS (N * NUMTHREADS * NUMBLOCKS)

__global__ void getHits (int *c) {    
    __shared__ int cache[NUMTHREADS];    
    int tid = threadIdx.x + blockIdx.x * blockDim.x;    
    int cacheIndex = threadIdx.x;
    
  // CUDA's random number library uses curandState_t to keep track of the seed value
  //   we will store a random state for every thread  
  hiprandState_t state;

  // we have to initialize the state 
  hiprand_init(tid, // the seed controls the sequence of random values that are produced 
              0, // the sequence number is only important with multiple cores 
              0, // the offset is how much extra we advance in the sequence for each call, can be 0 
              &state);

  int hits = 0;    
  double r, x, y;

    while (tid < NUMDARTS) {        
        // curand works like rand - except that it takes a state as a parameter 
        r = hiprand(&state) * 1.0 / (RAND_MAX); //  Between 0 and 1      
	x = -1 + 2 * r; //  Between -1 and 1
	r = hiprand(&state) * 1.0 / (RAND_MAX);
	y = -1 + 2 * r;
	if (((x * x) + (y * y)) <= 1)
	{
		hits++;
	}
        tid += blockDim.x * gridDim.x;    
    }        

    // set the cache values    
    cache[cacheIndex] = hits;

    __syncthreads();      

    // Reduction to sum up the results
    int i = blockDim.x/2;    
    while (i != 0) {        
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];    
        }    
        __syncthreads();        
        i /= 2;    
    }
    
    if (threadIdx.x == 0) {
	c[blockIdx.x] = cache[0]; 
    }
}

int main () {    
    int c[NUMBLOCKS]; //a[N], b[N],     
    int *dev_c;//[NUMBLOCKS]; //*dev_a, *dev_b, 
 
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // allocate the memory on the GPU    
    hipMalloc( (void**)&dev_c, NUMBLOCKS * sizeof(int) );
    
    hipEventRecord(start);
    // copy the arrays 'a' and 'b' to the GPU    
    getHits<<<NUMBLOCKS, NUMTHREADS>>>( dev_c );    
    hipEventRecord(stop);

    // copy the array 'c' back from the GPU to the CPU    
    hipMemcpy( c, dev_c, NUMBLOCKS * sizeof(int), hipMemcpyDeviceToHost );           
    double total_hits = 0.0;

    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Milliseconds, %f\n", milliseconds);

    int i = 0;
    while(i < NUMBLOCKS) {
	    total_hits += c[i];
	    i++;
    }

    printf("total_hits %f\n", total_hits);

    double pi_estimate = 16.0 * total_hits/NUMDARTS;
    printf("Estimate for pi:  %f \n", pi_estimate);
    
// free the memory allocated on the GPU    
    hipFree( dev_c );

    return 0; 
}
